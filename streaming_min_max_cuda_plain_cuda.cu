#include "hip/hip_runtime.h"
#include "utils.h"
#include "streaming_min_max_cuda_plain_cuda.cuh"

// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/**
 * CUDA kernel device code
 *
 * Computes a streaming minimum and maximum of the values of array \a
 * d_array within a window of size \a width and stores these these
 * minima and maxima in \a d_min and \a d_max respectively.
 */
__global__ void streamingMinMax(
    float const * d_array,
    float * d_min,
    float * d_max,
    unsigned int array_size,
    unsigned int min_max_size,
    unsigned int width
    )  
{
    int const thread_index(blockDim.x * blockIdx.x + threadIdx.x);
    float min, max;

    if (thread_index < min_max_size)
    {
	min = d_array[thread_index];
	max = d_array[thread_index];
	
	for (int i = 1; i < width; ++i)
	{
	    float current = d_array[thread_index + i];
	    
	    if (current < min)
	    {
		min = current;
	    }

	    if (current > max)
	    {
		max = current;
	    }
	}

	d_min[thread_index] = min;
	d_max[thread_index] = max;
    }
}


float * d_mem(NULL);

static void streaming_min_max_cuda_plain_clean_up(
    )
{
    hipError_t err(hipSuccess);

    if (d_mem != NULL)
    {
	TRACE(
	    "Freeing allocated device memory at 0x%lx ...\n",
	    (unsigned long) d_mem
	    );

	err = hipFree(d_mem);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to free allocated device memory at 0x%lx - %s",
		(unsigned long) d_mem,
		hipGetErrorString(err)
		);
	}
    }

    d_mem = NULL;
}

void streaming_min_max_cuda_plain_calc(
    float const * h_array,
    float * h_min,
    float * h_max,
    unsigned int array_size,
    unsigned int min_max_size,
    unsigned int width
    )
{
    unsigned int const total_mem_size((array_size + 2 * min_max_size) * sizeof(float));
    hipError_t err(hipSuccess);
    
    //
    // allocate device memory
    //

    TRACE(
	"Allocating %u bytes of device memory ...\n",
	total_mem_size
	);

    err = hipMalloc((void **) &d_mem, total_mem_size * sizeof(float));

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();

        ERROR_EXIT(
	    "Failed to allocate %d bytes of memory on device - %s",
	    total_mem_size,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Successfully allocated %u bytes of device memory at 0x%lx ...\n",
	total_mem_size,
	(unsigned long) d_mem
	);

    //
    // initialize pointers to subregions
    //

    float *d_array(d_mem);
    float *d_min(d_mem + array_size * sizeof(float));
    float *d_max(d_min + min_max_size * sizeof(float));

    //
    // copy input vector's data to device memory
    //

    TRACE(
	"Copying %u bytes of input data from vector 0x%lx into device memory 0x%lx ...\n",
	array_size,
	(unsigned long) h_array,
	(unsigned long) d_array
	);

    err = hipMemcpy(d_array, h_array, array_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %d bytes of input vector to device memory - %s",
	    array_size,
	    hipGetErrorString(err)
	    );
    }

    //
    // launch the CUDA kernel
    //

    int const threadsPerBlock(256);
    int const blocksPerGrid((array_size + threadsPerBlock - 1) / threadsPerBlock);

    TRACE(
	"Launching CUDA kernel with %d blocks of %d threads ...\n",
	blocksPerGrid,
	threadsPerBlock
	);   

    streamingMinMax<<<blocksPerGrid, threadsPerBlock>>>(
	d_array,
	d_min,
	d_max,
	array_size,
	min_max_size,
	width
	);    
    err = hipGetLastError();

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to launch kernel - %s",
	    hipGetErrorString(err)
	    );
    }

    //
    // copy output data from device memory into vectors
    //

    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_min,
	(unsigned long) h_min
	);

    err = hipMemcpy(h_min, d_min, min_max_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %d bytes of input vector to device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_max,
	(unsigned long) h_max
	);

    err = hipMemcpy(h_max, d_max, min_max_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %d bytes of input vector to device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
    }

    streaming_min_max_cuda_plain_clean_up();
}
