#include "hip/hip_runtime.h"
#include "utils.h"
#include "streaming_min_max_cuda_plain_cuda.cuh"

// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/**
 * CUDA kernel device code
 *
 * Computes a streaming minimum and maximum of the values of array \a
 * d_array within a window of size \a width and stores these these
 * minima and maxima in \a d_min and \a d_max respectively.
 */
__global__ void streamingMinMax(
    float const * d_array,
    float * d_min,
    float * d_max,
    unsigned int min_max_elements,
    
    unsigned int width
    )  
{
    int const thread_index(blockDim.x * blockIdx.x + threadIdx.x);
    float min, max;

    if (thread_index < min_max_elements)
    {
	min = d_array[thread_index];
	max = d_array[thread_index];
	
	for (int i = 1; i < width; ++i)
	{
	    float current = d_array[thread_index + i];
	    
	    if (current < min)
	    {
		min = current;
	    }

	    if (current > max)
	    {
		max = current;
	    }
	}

	d_min[thread_index] = min;
	d_max[thread_index] = max;
    }
}


static float *d_array(nullptr);
static float *d_min(nullptr);
static float *d_max(nullptr);

static float const *gh_array(nullptr);
static float *gh_min(nullptr);
static float *gh_max(nullptr);

static void streaming_min_max_cuda_plain_clean_up(
    )
{
    hipError_t err(hipSuccess);

    if (d_array != nullptr)
    {
	TRACE(
	    "Unregistering host memory at 0x%lx ...\n",
	    (unsigned long) gh_array
	    );

	err = hipHostUnregister((void *) gh_array);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to unregister memory at 0x%lx - %s",
		(unsigned long) gh_array,
		hipGetErrorString(err)
		);
	}
    }

    d_array = nullptr;

    if (d_min != nullptr)
    {
	TRACE(
	    "Unregistering host memory at 0x%lx ...\n",
	    (unsigned long) gh_min
	    );

	err = hipHostUnregister(gh_min);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to unregister memory at 0x%lx - %s",
		(unsigned long) gh_min,
		hipGetErrorString(err)
		);
	}
    }

    d_min = nullptr;

    if (d_max != nullptr)
    {
	TRACE(
	    "Unregistering host memory at 0x%lx ...\n",
	    (unsigned long) gh_max
	    );

	err = hipHostUnregister(gh_max);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to unregister memory at 0x%lx - %s",
		(unsigned long) gh_max,
		hipGetErrorString(err)
		);
	}
    }

    d_max = nullptr;
}

static void register_host_memory(
    float const * h_mem,
    float * &d_mem,
    unsigned int size
    )
{
    hipError_t err(hipSuccess);

    TRACE(
	"Registering %u bytes of host memory at 0x%lx for use by CUDA ...\n",
	size,
	(unsigned long) h_mem
	);

    err = hipHostRegister((void *) h_mem, size, hipHostRegisterMapped);
    
    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();

        ERROR_EXIT(
	    "Failed to register %u bytes of host memory at 0x%lx for use with CUDA - %s",
	    size,
	    (unsigned long) h_mem,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Successfully registered %u bytes of host memory at 0x%lx for use with CUDA ...\n",
	size,
	(unsigned long) h_mem
	);

    TRACE(
	"Obtaining device pointer for %u bytes of host memory at 0x%lx  ...\n",
	size,
	(unsigned long) h_mem
	);

    err = hipHostGetDevicePointer(&d_mem, (void *) h_mem, 0);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();

        ERROR_EXIT(
	    "Failed to obtain device pointer for %u bytes of host memory at 0x%lx - %s",
	    size,
  	    (unsigned long) h_mem,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Successfully obtained device pointer 0x%lx for %u bytes of host memory at 0x%lx ...\n",
	(unsigned long) d_mem,
	size,
	 (unsigned long) h_mem
	);
}

void streaming_min_max_cuda_plain_calc(
    float const * h_array,
    float * h_min,
    float * h_max,
    unsigned int array_elements,
    unsigned int min_max_elements,
    unsigned int width
    )
{
    unsigned int const min_max_size(min_max_elements * sizeof(float));
    unsigned int const array_size(array_elements * sizeof(float));
    hipError_t err(hipSuccess);
    int dev_count(0);
    hipDeviceProp_t dev_prop;

    //
    // query device properties
    //

    (void) hipGetDeviceCount(&dev_count);
    (void) hipGetDeviceProperties(&dev_prop, 0);

    TRACE(
	"Found %d devices and queried the following properties for device %d ...\n"
	"\tName: %s\n"
	"\tGlobal memory [bytes]: %u\n"
	"\tShared memory per block [bytes]: %u\n"
	"\tRegisters per block: %u\n"
	"\tWarp size: %u\n"
	"\tMaximum threads per block: %u\n"
	"\tCan map host memory: %s\n",
	dev_count,
	0,
	dev_prop.name,
	dev_prop.totalGlobalMem,
	dev_prop.sharedMemPerBlock,
	dev_prop.regsPerBlock,
	dev_prop.warpSize,
	dev_prop.maxThreadsPerBlock,
	(dev_prop.canMapHostMemory == 0) ? "no": "yes"
	);   
    
    int const threadsPerBlock(dev_prop.maxThreadsPerBlock);
    int const blocksPerGrid((array_size + threadsPerBlock - 1) / threadsPerBlock);

    //
    // register host memory with device
    //

    register_host_memory(
	h_array,
	d_array,
	array_size
	);
    gh_array = h_array;
    
    register_host_memory(
	h_min,
	d_min,
	min_max_size
	);
    gh_min = h_min;

    register_host_memory(
	h_max,
	d_max,
	min_max_size
	);
    gh_max = h_max;

    //
    // launch the CUDA kernel
    //

    TRACE(
	"Launching CUDA kernel with %d blocks of %d threads ...\n",
	blocksPerGrid,
	threadsPerBlock
	);   

    streamingMinMax<<<blocksPerGrid, threadsPerBlock>>>(
	d_array,
	d_min,
	d_max,
	min_max_elements,
	width
	);    
    err = hipGetLastError();

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to launch kernel - %s",
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Waiting for CUDA kernel to finish ...\n"
	);   

    err = hipDeviceSynchronize();
    
    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to wait for CUDA kernel to finish - %s",
	    hipGetErrorString(err)
	    );
    }

    streaming_min_max_cuda_plain_clean_up();
}
