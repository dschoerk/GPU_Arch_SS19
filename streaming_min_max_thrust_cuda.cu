#include "hip/hip_runtime.h"
#include "streaming_min_max_cuda_plain_cuda.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>

// alias dptr = thrust::device_ptr<float>;

struct ShiftedWindowMinMaxStep
{
    int win_size;
    thrust::device_ptr<float> data;
    thrust::device_ptr<float> out_max;
    thrust::device_ptr<float> out_min;
    
    ShiftedWindowMinMaxStep(
        thrust::device_ptr<float> _data, 
        int _win_size, 
        thrust::device_ptr<float> _out_min, 
        thrust::device_ptr<float> _out_max
    ) : data(_data), win_size(_win_size), out_min(_out_min), out_max(_out_max) {}

    __device__ void operator() (const int &idx)
    {
        // we use the parameter as index from a counting iterator
        // we store the min/max index

        float min = data[idx];
        float max = min;

        //printf("data %d: %f\n", idx, min);

        for(int i = 1; i < win_size; ++i)
        {
            float e = data[idx + i];
            min = fminf(min, e);
            max = fmaxf(max, e);
            
            // __syncthreads();
        }

        out_min[idx] = min;        
        out_max[idx] = max;

        //out_min[idx] = idx;
    }
};

struct ShiftedWindowMinMaxStep2
{
    int win_size;
    int data_size;
    thrust::device_ptr<float> data;
    thrust::device_ptr<float> out_max;
    thrust::device_ptr<float> out_min;
    thrust::device_ptr<float> tmp_max;
    thrust::device_ptr<float> tmp_min;

    int win_size_log2; // next smaller power of 2
    
    ShiftedWindowMinMaxStep2(
        thrust::device_ptr<float> _data, 
        int _win_size, 
        int _data_size,
        thrust::device_ptr<float> _tmp_min, 
        thrust::device_ptr<float> _tmp_max,
        thrust::device_ptr<float> _out_min, 
        thrust::device_ptr<float> _out_max
    ) : data(_data), win_size(_win_size), out_min(_out_min), out_max(_out_max), tmp_min(_tmp_min), tmp_max(_tmp_max) , data_size(_data_size)
    {
        for(win_size_log2 = 0; 1 << win_size_log2 <= win_size; ++win_size_log2); // find next smaller power of 2 to win_size
        win_size_log2--;

        printf("win_size_log2: %d\n", win_size_log2);
        printf("summation steps: %d\n", win_size - (1 << win_size_log2));
    }

    __device__ void operator() (const int &k)
    {
        //if (k > 250)
        /*{
            printf("v: %d %f\n", k, data[k]);
        }*/
        
        for(int d = 0; d < win_size_log2; ++d) // naive and incomplete prefix sum
        {
            if(k + (1 << d) < data_size)
            {
                tmp_min[k] = fminf(tmp_min[k], tmp_min[k + (1 << d)]);
                tmp_max[k] = fmaxf(tmp_max[k], tmp_max[k + (1 << d)]);
            }

            __syncthreads();
        }

        if(k < data_size - win_size + 1) 
        {
            out_min[k] = tmp_min[k];
            out_max[k] = tmp_max[k];

            __syncthreads();

            for(int i = 0; i < win_size - (1 << win_size_log2); ++i) // sum up the rest
            {
                out_min[k] = fminf(out_min[k], tmp_min[k+i+1]);
                out_max[k] = fmaxf(out_max[k], tmp_max[k+i+1]);

                __syncthreads();
            }
        }
    }
};

void streaming_min_max_thrust_calc(
    std::vector<float> const & array,
    unsigned int width,
    std::vector<float> & minvalues,
    std::vector<float> & maxvalues
)
{
    //thrust::host_vector<float> h_vec(array);

    // transfer data to the device
    thrust::device_vector<float> d_vec(array); // = h_vec;
    thrust::device_vector<float> d_minima(d_vec.size());
    thrust::device_vector<float> d_maxima(d_vec.size());

    thrust::device_vector<float> d_tmp_minima(array);
    thrust::device_vector<float> d_tmp_maxima(array);
    

    
    // https://stackoverflow.com/questions/21761412/thrust-reduction-result-on-device-memory
    // thrust reductions return to host memory
    // we need to use reduce by key to keep it on host memory
    // otherwise we copy every min/max value from device to host separately -> super slow


    // stupid and simple version
    /*
    int i = 0;
    for(auto it = d_vec.begin(); it + width - 1 != d_vec.end(); ++it)
    {
        auto result = thrust::minmax_element(d_vec.begin() + i, d_vec.begin() + i + width);
        
        minvalues[i] = *result.first;
        maxvalues[i] = *result.second;
        ++i;
    }*/

    thrust::counting_iterator<int> c_begin(0);
    //thrust::counting_iterator<int> c_end(d_vec.size() - width + 1); // inclusive end?
    thrust::counting_iterator<int> c_end(d_vec.size());

    //for all shifts
    {
        thrust::for_each(c_begin, c_end,
            ShiftedWindowMinMaxStep2(
                d_vec.data(), 
                width, 
                array.size(),
                d_tmp_minima.data(), 
                d_tmp_maxima.data(),
                d_minima.data(), 
                d_maxima.data()
            ));
    }

    thrust::copy(d_minima.begin(), d_minima.end() - width + 1, minvalues.begin()); // is this efficient?
    thrust::copy(d_maxima.begin(), d_maxima.end() - width + 1, maxvalues.begin());
    
    /*thrust::host_vector<float> h_minima(minvalues.begin(), minvalues.end());
    thrust::host_vector<float> h_maxima(maxvalues.begin(), maxvalues.end());

    h_minima = d_minima;
    h_maxima = d_maxima;*/


    for(int i=505; i < 514; i++)
        std::cout << i << ": " << array[i] << ", ";

    /*for(float f : thrust::host_vector<float>(d_vec))
        std::cout << f << ", ";
    std::cout << std::endl << std::endl;

    for(float f : minvalues )
        std::cout << f << ", ";
    std::cout << std::endl << std::endl;

    for(float f : maxvalues)
        std::cout << f << ", ";
    std::cout << std::endl << std::endl;*/

}