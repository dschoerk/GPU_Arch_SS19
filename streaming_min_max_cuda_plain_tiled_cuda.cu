#include "hip/hip_runtime.h"

#include "utils.h"
#include "streaming_min_max_cuda_plain_tiled_cuda.cuh"

// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <nvToolsExt.h>

#include <hip/hip_cooperative_groups.h>

#define BLOCK_SIZE 1024

namespace cg = cooperative_groups;

/**
 * CUDA kernel device code
 *
 * Computes a streaming minimum and maximum of the values of array \a
 * d_array within a window of size \a width and stores these these
 * minima and maxima in \a d_min and \a d_max respectively.
 */
__global__ void streaming_min_max_cuda_plain_tiled_calc(
    float const * d_array,
    float * d_min,
	float * d_max,
	const unsigned int array_elements,
    unsigned int min_max_elements,
	unsigned int win_size,
	unsigned int win_size_log2
    )  
{


	// tiling by blocks
	// unsigned int tile = blockIdx.x;
	// unsigned int tile_offset = blockDim.x * blockIdx.x;

	const int BLOCK_IDX = blockIdx.x; 
	const int OUT_BLOCK_SIZE = BLOCK_SIZE - win_size + 1; // the number of results a block can compute
	const int k = threadIdx.x;

	//if(tile_offset == 0 && k == 0)
	//	printf("OUT_BLOCK_SIZE: %d BLOCK_SIZE: %d win_size: %d 2^win_size_log2: %d\n", OUT_BLOCK_SIZE, BLOCK_SIZE, win_size, (1<<win_size_log2));

	
	//if(BLOCK_IDX > 0)
	//	return;

	if(k >= array_elements)
		return;

	__shared__ float s_min[BLOCK_SIZE];
	__shared__ float s_max[BLOCK_SIZE];


	if( k + BLOCK_IDX * OUT_BLOCK_SIZE < array_elements )
	{
		float read = d_array[k + BLOCK_IDX * OUT_BLOCK_SIZE];
		s_min[k] = read;
		s_max[k] = read;
	}

	__syncthreads();
	
	for(int d = 0; d < win_size_log2; ++d) // naive and incomplete prefix sum
	{
		// pack this into separate kernels
		float min1, min2, max1, max2;
		
		if(k + (1 << d) < BLOCK_SIZE)
		{
			min1 = s_min[k];
			min2 = s_min[k + (1 << d)];
			max1 = s_max[k];
			max2 = s_max[k + (1 << d)];
		}
			
		__syncthreads();
		
		if(k + (1 << d) < BLOCK_SIZE)
		{
			if(min2 < min1)
				s_min[k] = min2;

			if(max2 > max1)
				s_max[k] = max2;
		}

			//printf("writing %d %f %f\n", k, min1, min2);
			
		__syncthreads();
	}

	if(OUT_BLOCK_SIZE * BLOCK_IDX + k < array_elements - win_size + 1 && k < OUT_BLOCK_SIZE)
	//if( k < )
	{
		float min = s_min[k];
		float max = s_max[k];

		__syncthreads();

		for(int i = 0; i < win_size - (1 << win_size_log2); ++i) // sum up the rest
		{
			min = fminf(min, s_min[k+i+1]);
			max = fmaxf(max, s_max[k+i+1]);

			__syncthreads();
		}

		//printf("out: %d blk: %d k: %d min: %f max: %f\n", OUT_BLOCK_SIZE * BLOCK_IDX + k, BLOCK_IDX, k, min, max);

		d_min[OUT_BLOCK_SIZE * BLOCK_IDX + k] = min;
		d_max[OUT_BLOCK_SIZE * BLOCK_IDX + k] = max;
	}
	
}


float * d_mem_tiled(NULL);

static void streaming_min_max_cuda_plain_clean_up(
    )
{
    hipError_t err(hipSuccess);

    if (d_mem_tiled != NULL)
    {
	TRACE(
	    "Freeing allocated device memory at 0x%lx ...\n",
	    (unsigned long) d_mem_tiled
	    );

	err = hipFree(d_mem_tiled);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to free allocated device memory at 0x%lx - %s",
		(unsigned long) d_mem_tiled,
		hipGetErrorString(err)
		);
	}
    }

    d_mem_tiled = NULL;
}

void streaming_min_max_cuda_plain_tiled_calc(
    float const * h_array,
    float * h_min,
    float * h_max,
    unsigned int array_elements,
    unsigned int min_max_elements,
    unsigned int width
    )
{
PUSH_RANGE("h2d", 1)
	
    unsigned int const min_max_size = min_max_elements * sizeof(float);
    unsigned int const array_size = array_elements * sizeof(float);
    unsigned int const total_mem_size(array_size + 2 * min_max_size);
    hipError_t err(hipSuccess);    
    int dev_count(0);
    hipDeviceProp_t dev_prop;

    //
    // query device properties
    //

    (void) hipGetDeviceCount(&dev_count);
    (void) hipGetDeviceProperties(&dev_prop, 0);

    TRACE(
	"Found %d devices and queried the following properties for device %d ...\n"
	"\tName: %s\n"
	"\tGlobal memory [bytes]: %u\n"
	"\tShared memory per block [bytes]: %u\n"
	"\tRegisters per block: %u\n"
	"\tWarp size: %u\n"
	"\tMaximum threads per block: %u\n"
	"\tCooperative Launch: %d\n"
	"\tCompute Capability: %d.%d\n"
	"\tCan map host memory: %s\n",
	dev_count,
	0,
	dev_prop.name,
	dev_prop.totalGlobalMem,
	dev_prop.sharedMemPerBlock,
	dev_prop.regsPerBlock,
	dev_prop.warpSize,
	dev_prop.maxThreadsPerBlock,
	dev_prop.cooperativeLaunch,
	dev_prop.major, dev_prop.minor,
	(dev_prop.canMapHostMemory == 0) ? "no": "yes"
	);   

	int win_size_log2;
    for(win_size_log2 = 0; 1 << win_size_log2 <= width; ++win_size_log2); // find next smaller power of 2 to win_size
	win_size_log2--;
	

	const int OUT_BLOCK_SIZE = BLOCK_SIZE - width + 1;
	int const threadsPerBlock = BLOCK_SIZE; //dev_prop.maxThreadsPerBlock);
	
	const int output_blocks = min_max_elements / OUT_BLOCK_SIZE + OUT_BLOCK_SIZE;

	int const blocksPerGrid = output_blocks;//(((output_blocks * threadsPerBlock) + threadsPerBlock - 1) / threadsPerBlock);
	
    //
    // allocate device memory
    //

    TRACE(
	"Allocating %u bytes of device memory ...\n",
	total_mem_size
	);

    err = hipMalloc((void **) &d_mem_tiled, total_mem_size);

    if (err != hipSuccess)
    {
		streaming_min_max_cuda_plain_clean_up();

        ERROR_EXIT(
	    "Failed to allocate %u bytes of memory on device - %s",
	    total_mem_size,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Successfully allocated %u bytes of device memory at 0x%lx ...\n",
	total_mem_size,
	(unsigned long) d_mem_tiled
	);

    //
    // initialize pointers to subregions
    //

    float *d_array(d_mem_tiled);
    float *d_min(d_mem_tiled + array_elements);
    float *d_max(d_min + min_max_elements);

    //
    // copy input vector's data to device memory
    //

    TRACE(
	"Copying %u bytes of input data from vector 0x%lx into device memory 0x%lx ...\n",
	array_size,
	(unsigned long) h_array,
	(unsigned long) d_array
	);

    err = hipMemcpy(d_array, h_array, array_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes of input vector to device memory - %s",
	    array_size,
	    hipGetErrorString(err)
	    );
    }

    //
    // launch the CUDA kernel
    //

    TRACE(
	"Launching CUDA kernel with %d blocks of %d threads ...\n",
	blocksPerGrid,
	threadsPerBlock
	);   

	

POP_RANGE

PUSH_RANGE("kernel", 2)

    streaming_min_max_cuda_plain_tiled_calc<<<blocksPerGrid, threadsPerBlock>>>(
	d_array,
	d_min,
	d_max,
	array_elements,
	min_max_elements,
	width,
	win_size_log2
	);   

hipDeviceSynchronize();
POP_RANGE

PUSH_RANGE("d2h", 3)
    err = hipGetLastError();

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to launch kernel - %s",
	    hipGetErrorString(err)
	    );
    }

    //
    // copy output data from device memory into vectors
    //

	TRACE("check1\n");
    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_min,
	(unsigned long) h_min
	);

	TRACE("hipMemcpy %d\n", min_max_size);
	err = hipMemcpy(h_min, d_min, min_max_size, hipMemcpyDeviceToHost);
	TRACE("check2\n");

    if (err != hipSuccess)
    {
		streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes to input vector from device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
	}
	
	TRACE("check3");

    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_max,
	(unsigned long) h_max
	);

    err = hipMemcpy(h_max, d_max, min_max_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes to input vector from device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
    }

	streaming_min_max_cuda_plain_clean_up();
	
POP_RANGE
}
