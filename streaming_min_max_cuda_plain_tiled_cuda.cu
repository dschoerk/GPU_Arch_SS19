#include "hip/hip_runtime.h"

#include "utils.h"
#include "streaming_min_max_cuda_plain_tiled_cuda.cuh"

// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <nvToolsExt.h>

#include <hip/hip_cooperative_groups.h>

#define BLOCK_SIZE 1024

namespace cg = cooperative_groups;

/**
 * CUDA kernel device code
 *
 * Computes a streaming minimum and maximum of the values of array \a
 * d_array within a window of size \a width and stores these these
 * minima and maxima in \a d_min and \a d_max respectively.
 */
__global__ void streaming_min_max_cuda_plain_tiled_calc(
    float const * d_array,
    float * d_min,
	float * d_max,
	const unsigned int array_elements,
    unsigned int min_max_elements,
	unsigned int win_size,
	unsigned int win_size_log2
    )  
{
	// tiling by blocks
	unsigned int tile = blockIdx.x;
	unsigned int tile_offset = blockDim.x * blockIdx.x;

	//int const thread_index(blockDim.x * blockIdx.x + threadIdx.x);

	const int k = threadIdx.x;

	if(k >= array_elements)
		return;

	__shared__ float s_min[BLOCK_SIZE];
	__shared__ float s_max[BLOCK_SIZE];


	if( k < array_elements )
	{
		/*d_min[k] = d_array[k];
		d_max[k] = d_array[k];*/
		
		s_min[k] = d_array[k];
		s_max[k] = d_array[k];
		

		//printf("%d = %f\n", k, d_min[k]);
	}

	__syncthreads();
	
	for(int d = 0; d < win_size_log2; ++d) // naive and incomplete prefix sum
	{
		// pack this into separate kernels
		float min1, min2, max1, max2;
		
		if(k + (1 << d) < array_elements)
		{
			min1 = s_min[k];
			min2 = s_min[k + (1 << d)];
			max1 = s_max[k];
			max2 = s_max[k + (1 << d)];
		}
			
		__syncthreads();
		
		if(k + (1 << d) < array_elements)
		{
			if(min2 < min1)
				s_min[k] = min2;

			if(max2 > max1)
				s_max[k] = max2;
		}

			//printf("writing %d %f %f\n", k, min1, min2);
			
		__syncthreads();
	}


	/*return;

	if(k == 0)
	{
		printf("win_size: %d (1 << win_size_log2): %d\n", win_size, (1 << win_size_log2));
	}*/

	// for non power of 2's
	//if(k < array_elements) 

	if(k < array_elements - win_size + 1)
	{
		float min = s_min[k];
		float max = s_max[k];

		__syncthreads();

		for(int i = 0; i < win_size - (1 << win_size_log2); ++i) // sum up the rest
		{
			min = fminf(min, s_min[k+i+1]);
			max = fmaxf(max, s_max[k+i+1]);

			__syncthreads();
		}

		d_min[k] = min;
		d_max[k] = max;
	}
	


    /*float min, max;

    if (thread_index < min_max_elements)
    {
		min = d_array[thread_index];
		max = d_array[thread_index];
		
		for (int i = 1; i < width; ++i)
		{
			float current = d_array[thread_index + i];
			
			if (current < min)
			{
				min = current;
			}

			if (current > max)
			{
				max = current;
			}
		}

		d_min[thread_index] = min;
		d_max[thread_index] = max;
    }*/
}


float * d_mem_tiled(NULL);

static void streaming_min_max_cuda_plain_clean_up(
    )
{
    hipError_t err(hipSuccess);

    if (d_mem_tiled != NULL)
    {
	TRACE(
	    "Freeing allocated device memory at 0x%lx ...\n",
	    (unsigned long) d_mem_tiled
	    );

	err = hipFree(d_mem_tiled);

	if (err != hipSuccess)
	{
	    ERROR_EXIT(
		"Failed to free allocated device memory at 0x%lx - %s",
		(unsigned long) d_mem_tiled,
		hipGetErrorString(err)
		);
	}
    }

    d_mem_tiled = NULL;
}

void streaming_min_max_cuda_plain_tiled_calc(
    float const * h_array,
    float * h_min,
    float * h_max,
    unsigned int array_elements,
    unsigned int min_max_elements,
    unsigned int width
    )
{
	nvtxRangePushA("prepare (h2d)");
    unsigned int const min_max_size = min_max_elements * sizeof(float);
    unsigned int const array_size = array_elements * sizeof(float);
    unsigned int const total_mem_size(array_size + 2 * array_size);
    hipError_t err(hipSuccess);    
    int dev_count(0);
    hipDeviceProp_t dev_prop;

    //
    // query device properties
    //

    (void) hipGetDeviceCount(&dev_count);
    (void) hipGetDeviceProperties(&dev_prop, 0);

    TRACE(
	"Found %d devices and queried the following properties for device %d ...\n"
	"\tName: %s\n"
	"\tGlobal memory [bytes]: %u\n"
	"\tShared memory per block [bytes]: %u\n"
	"\tRegisters per block: %u\n"
	"\tWarp size: %u\n"
	"\tMaximum threads per block: %u\n"
	"\tCooperative Launch: %d\n"
	"\tCompute Capability: %d.%d\n"
	"\tCan map host memory: %s\n",
	dev_count,
	0,
	dev_prop.name,
	dev_prop.totalGlobalMem,
	dev_prop.sharedMemPerBlock,
	dev_prop.regsPerBlock,
	dev_prop.warpSize,
	dev_prop.maxThreadsPerBlock,
	dev_prop.cooperativeLaunch,
	dev_prop.major, dev_prop.minor,
	(dev_prop.canMapHostMemory == 0) ? "no": "yes"
	);   

    int const threadsPerBlock(dev_prop.maxThreadsPerBlock);
	int const blocksPerGrid((array_elements + threadsPerBlock - 1) / threadsPerBlock);
	
    //
    // allocate device memory
    //

    TRACE(
	"Allocating %u bytes of device memory ...\n",
	total_mem_size
	);

    err = hipMalloc((void **) &d_mem_tiled, total_mem_size);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();

        ERROR_EXIT(
	    "Failed to allocate %u bytes of memory on device - %s",
	    total_mem_size,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Successfully allocated %u bytes of device memory at 0x%lx ...\n",
	total_mem_size,
	(unsigned long) d_mem_tiled
	);

    //
    // initialize pointers to subregions
    //

    float *d_array(d_mem_tiled);
    float *d_min(d_mem_tiled + array_elements);
    float *d_max(d_min + array_elements);

    //
    // copy input vector's data to device memory
    //

    TRACE(
	"Copying %u bytes of input data from vector 0x%lx into device memory 0x%lx ...\n",
	array_size,
	(unsigned long) h_array,
	(unsigned long) d_array
	);

    err = hipMemcpy(d_array, h_array, array_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes of input vector to device memory - %s",
	    array_size,
	    hipGetErrorString(err)
	    );
    }

    //
    // launch the CUDA kernel
    //

    TRACE(
	"Launching CUDA kernel with %d blocks of %d threads ...\n",
	blocksPerGrid,
	threadsPerBlock
	);   

	int win_size_log2;
    for(win_size_log2 = 0; 1 << win_size_log2 <= width; ++win_size_log2); // find next smaller power of 2 to win_size
    win_size_log2--;

	nvtxRangePop();

	nvtxRangePushA("compute");
    streaming_min_max_cuda_plain_tiled_calc<<<blocksPerGrid, threadsPerBlock>>>(
	d_array,
	d_min,
	d_max,
	array_elements,
	min_max_elements,
	width,
	win_size_log2
	);    
	nvtxRangePop();

	nvtxRangePushA("prepare (h2d)");
    err = hipGetLastError();

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to launch kernel - %s",
	    hipGetErrorString(err)
	    );
    }

    //
    // copy output data from device memory into vectors
    //

    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_min,
	(unsigned long) h_min
	);

    err = hipMemcpy(h_min, d_min, min_max_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes of input vector to device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
    }

    TRACE(
	"Copying %u bytes of output data from device memory 0x%lx into vector 0x%lx ...\n",
	min_max_size,
	(unsigned long) d_max,
	(unsigned long) h_max
	);

    err = hipMemcpy(h_max, d_max, min_max_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
	streaming_min_max_cuda_plain_clean_up();
	
        ERROR_EXIT(
	    "Failed to copy %u bytes of input vector from device memory - %s",
	    min_max_size,
	    hipGetErrorString(err)
	    );
    }

	streaming_min_max_cuda_plain_clean_up();
	
	nvtxRangePop();
}
